#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>
#include <string.h> 
#include <hip/hip_runtime.h>

#include "helper.cuh" 

#define GPU_RUNS 100 
#define N 753411
#define CPU_RUNS 


void naive_map(float *arr, int n, float *result, int result_size){
    if (result_size < n) {
        if (result_size < n){
            printf("The result is less than n\n");
            assert(0)
        }
        for (int i=0; i < n; i++){
            float x = arr[i];
            float temp = x / (x - 2.3);
            result[i] = temp * temp * temp; 
        }
    }
    __global__ void cuda_map(float* X, float* Y, int n){
        const unsigned
    }
}

